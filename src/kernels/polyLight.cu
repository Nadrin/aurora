#include "hip/hip_runtime.h"
/* Aurora Renderer
 * Copyright (c) 2013 Michal Siejak
 * Licensed under MIT open-source license, see COPYING.txt file for details.
 */

#include <stdafx.h>
#include <kernels/kernels.h>

#include <thrust/device_ptr.h>
#include <thrust/copy.h>
#include <thrust/scan.h>

using namespace Aurora;

#include <kernels/lib/common.cuh>
#include <kernels/lib/intersect.cuh>
#include <kernels/lib/bsdf.cuh>
#include <kernels/lib/shader.cuh>

struct ValidPolyLightPredicate
{
	__device__ bool operator()(const PolyLight& pl)
	{ return pl.emission > 0.0f; }
};

struct PolyLightCdfBinaryOp
{
	__device__ PolyLight operator()(const PolyLight& a, const PolyLight& b)
	{
		PolyLight result;
		result.emission   = b.emission;
		result.triangleID = b.triangleID;
		result.cdf        = a.emission + b.emission;
		return result;
	}
};

__global__ static void cudaInitPolyLights(const Geometry geometry, const ShadersArray shaders, PolyLight* lights)
{
	const unsigned int threadId = blockDim.x * blockIdx.x + threadIdx.x;
	if(threadId >= geometry.count)
		return;

	const unsigned int shaderID = getSafeID(geometry.shaders[threadId]);
	lights[threadId].triangleID = threadId;
	lights[threadId].emission   = shaders[shaderID].emission;
}

__global__ static void cudaCalculatePolyLightsCDF(const Geometry geometry, const unsigned int numLights, PolyLight* lights)
{
	const unsigned int threadId = blockDim.x * blockIdx.x + threadIdx.x;
	if(threadId >= numLights)
		return;

	volatile __shared__ float cdfIntegral;
	if(threadId == 0)
		cdfIntegral = lights[numLights-1].cdf;
	__syncthreads();
	lights[threadId].cdf /= cdfIntegral;

	Primitive3 triangle;
	triangle.readPoints(geometry.vertices + lights[threadId].triangleID * Geometry::TriangleParams);
	lights[threadId].area = triangle.area();
}

__host__ unsigned int cudaCreatePolyLights(const Geometry& geometry, const ShadersArray& shaders, PolyLight** lights)
{
	PolyLight* buffer;

	*lights = NULL;
	if(hipMalloc(&buffer, sizeof(PolyLight) * geometry.count) != hipSuccess)
		return 0;
	if(hipMalloc(lights, sizeof(PolyLight) * geometry.count) != hipSuccess) {
		hipFree(buffer);
		return 0;
	}

	dim3 blockSize;
	dim3 gridSize;

	blockSize = dim3(512);
	gridSize  = make_grid(blockSize, geometry.count);
	cudaInitPolyLights<<<gridSize, blockSize>>>(geometry, shaders, *lights);

	thrust::device_ptr<PolyLight> thrustBuffer(buffer);
	thrust::device_ptr<PolyLight> thrustLights(*lights);

	const auto thrustBufferEnd   = thrust::copy_if(thrustLights, thrustLights + geometry.count, thrustBuffer, ValidPolyLightPredicate());
	const unsigned int numLights = thrustBufferEnd - thrustBuffer;
	if(numLights == 0) {
		hipFree(buffer);
		return 0;
	}

	thrust::inclusive_scan(thrustBuffer, thrustBuffer + numLights, thrustLights, PolyLightCdfBinaryOp());
	hipFree(buffer);

	blockSize = dim3(256);
	gridSize  = make_grid(blockSize, numLights);
	cudaCalculatePolyLightsCDF<<<gridSize, blockSize>>>(geometry, numLights, *lights);
	return numLights;
}
