#include "hip/hip_runtime.h"
/* Aurora Renderer
 * Copyright (c) 2013 Michal Siejak
 * Licensed under MIT open-source license, see COPYING.txt file for details.
 */

#include <stdafx.h>
#include <kernels/kernels.h>

#include <thrust/device_ptr.h>
#include <thrust/copy.h>
#include <thrust/scan.h>

using namespace Aurora;

#include <kernels/lib/common.cuh>
#include <kernels/lib/intersect.cuh>
#include <kernels/lib/bsdf.cuh>
#include <kernels/lib/shader.cuh>

struct ValidEmitterPredicate
{
	__device__ bool operator()(const Emitter& e)
	{ return e.pdf > 0.0f; }
};

struct EmitterCdfBinaryOp
{
	__device__ Emitter operator()(const Emitter& a, const Emitter& b)
	{
		Emitter result;
		result.pdf        = b.pdf;
		result.power      = b.power;
		result.triangleID = b.triangleID;
		result.cdf        = a.cdf + b.cdf;
		return result;
	}
};

__global__ static void cudaInitEmitters(const Geometry geometry, const Shader* shaders, Emitter* emitters)
{
	const unsigned int threadId = blockDim.x * blockIdx.x + threadIdx.x;
	if(threadId >= geometry.count)
		return;

	const unsigned int shaderID = getSafeID(geometry.shaders[threadId]);

	emitters[threadId].power      = shaders[shaderID].emissionColor;
	emitters[threadId].pdf        = shaders[shaderID].emission;
	emitters[threadId].triangleID = threadId;
}

__global__ static void cudaCalculateEmittersCDF(const Geometry geometry,
	const unsigned int numEmitters, Emitter* lights)
{
	const unsigned int threadId = blockDim.x * blockIdx.x + threadIdx.x;
	if(threadId >= numEmitters)
		return;

	Primitive3 triangle;
	triangle.readPoints(geometry.vertices + lights[threadId].triangleID * Geometry::TriangleParams);
	
	const float area = triangle.area();
	lights[threadId].area = area;
	lights[threadId].cdf  = lights[threadId].pdf; // * area;
	lights[threadId].pdf  = 1.0f / area;
}

__global__ static void cudaNormalizeEmittersCDF(const unsigned int numEmitters, Emitter* lights,
	const float cdfIntegral)
{
	const unsigned int threadId = blockDim.x * blockIdx.x + threadIdx.x;
	if(threadId >= numEmitters)
		return;

	//lights[threadId].pdf /= (cdfIntegral * numEmitters);
	lights[threadId].cdf /= cdfIntegral;
}

unsigned int cudaCreateEmitters(const Geometry& geometry, const ShadersArray& shaders, Emitter** emitters)
{
	Emitter* buffer;

	*emitters = NULL;
	if(hipMalloc(&buffer, sizeof(Emitter) * geometry.count) != hipSuccess)
		return 0;
	if(hipMalloc(emitters, sizeof(Emitter) * geometry.count) != hipSuccess) {
		hipFree(buffer);
		return 0;
	}

	dim3 blockSize = dim3(256);
	dim3 gridSize;

	gridSize  = make_grid(blockSize, dim3(geometry.count));
	cudaInitEmitters<<<gridSize, blockSize>>>(geometry, shaders.items, *emitters);

	thrust::device_ptr<Emitter> thrustBuffer(buffer);
	thrust::device_ptr<Emitter> thrustEmitters(*emitters);

	const auto thrustBufferEnd     = thrust::copy_if(thrustEmitters, thrustEmitters + geometry.count, thrustBuffer, ValidEmitterPredicate());
	const unsigned int numEmitters = thrustBufferEnd - thrustBuffer;
	if(numEmitters == 0) {
		hipFree(buffer);
		return 0;
	}

	gridSize  = make_grid(blockSize, dim3(numEmitters));
	cudaCalculateEmittersCDF<<<gridSize, blockSize>>>(geometry, numEmitters, buffer);

	thrust::inclusive_scan(thrustBuffer, thrustBuffer + numEmitters, thrustEmitters, EmitterCdfBinaryOp());
	hipFree(buffer);

	Emitter lastElement;
	hipMemcpy(&lastElement, (*emitters) + (numEmitters-1), sizeof(Emitter), hipMemcpyDeviceToHost);

	gridSize  = make_grid(blockSize, dim3(numEmitters));
	cudaNormalizeEmittersCDF<<<gridSize, blockSize>>>(numEmitters, *emitters, lastElement.cdf);
	return numEmitters;
}
