#include "hip/hip_runtime.h"
/* Aurora Renderer
 * Copyright (c) 2013 Michal Siejak
 * Licensed under MIT open-source license, see COPYING.txt file for details.
 */

#include <stdafx.h>
#include <kernels/kernels.h>

#include <thrust/sequence.h>
#include <thrust/device_vector.h>
#include <thrust/gather.h>
#include <thrust/sort.h>

using namespace Aurora;

#include <kernels/common.h>

#define REBUILDER_TPS_THRESHOLD 2048

struct Split 
{
	__host__ __device__ 
	Split() : index(0), size(0) { }

	__host__ __device__ 
	Split(const unsigned int _index, const unsigned int _size) :
		index(_index), size(_size) { }

	unsigned int index;
	unsigned int size;
};

struct ActiveSplitPredicate
{
	__device__ bool operator()(const Split& split)
	{ return split.size > 0; }
};

inline __device__ void calcPartition(const unsigned int N, unsigned int& L, unsigned int& R)
{
	const unsigned int n = N / 2;
	const unsigned int H = log2i(n);
	const unsigned int s = exp2i(H-1) - 1;
	const unsigned int S = exp2i(H) - 1;
	const unsigned int O = max(0, int((n-1) - s - S));

	R = 2 * (s+O);
	L = 2 * (n-1) - R;
}

inline __device__ unsigned int findSplit(const unsigned int index, const unsigned int pendingSplits,
	const Split* splitArray, Split& outSplit)
{
	unsigned int imin = 0;
	unsigned int imax = pendingSplits;
	unsigned int outIndex;

	while(imax >= imin) {
		outIndex = (imin + imax) / 2;
		outSplit = splitArray[outIndex];

		if(index < outSplit.index)
			imax = outIndex - 1;
		else if(index > outSplit.index && index >= outSplit.index + outSplit.size)
			imin = outIndex + 1;
		else break;
	}
	return outIndex;
}

__global__ static void computeTriangleBounds(const Geometry geometry,
	float* verticesMinX, float* verticesMinY, float* verticesMinZ,
	float* verticesMaxX, float* verticesMaxY, float* verticesMaxZ)
{
	const unsigned int index = blockDim.x * blockIdx.x + threadIdx.x;
	if(index >= geometry.count)
		return;

	Primitive vertices;
	vertices.readPoints(geometry.vertices + index * Geometry::TriangleParams);

	verticesMinX[index] = fminf(fminf(vertices.v1.x, vertices.v2.x), vertices.v3.x);
	verticesMaxX[index] = fmaxf(fmaxf(vertices.v1.x, vertices.v2.x), vertices.v3.x);

	verticesMinY[index] = fminf(fminf(vertices.v1.y, vertices.v2.y), vertices.v3.y);
	verticesMaxY[index] = fmaxf(fmaxf(vertices.v1.y, vertices.v2.y), vertices.v3.y);

	verticesMinZ[index] = fminf(fminf(vertices.v1.z, vertices.v2.z), vertices.v3.z);
	verticesMaxZ[index] = fmaxf(fmaxf(vertices.v1.z, vertices.v2.z), vertices.v3.z);
}

__global__ static void swapMaxTriangleTPT(const unsigned int count, const float* keys,
	const unsigned int pendingSplits, const Split* splits, unsigned int* indices,
	const unsigned int doneNodes, unsigned int* nodes)
{
	unsigned int index = blockDim.x * blockIdx.x + threadIdx.x;
	if(index >= count - 2*doneNodes)
		return;
	index = count - index - 1;

	Split split;
	findSplit(index, pendingSplits, splits, split);

	if(index == split.index || index == split.index+1)
		return;

	unsigned int threadIndex = indices[index];
	float threadValue        = keys[threadIndex];

	while(true) {
		unsigned int currentIndex = indices[split.index+1];
		float currentValue        = keys[currentIndex];

		if(threadValue > currentValue) {
			if(atomicCAS(&indices[split.index+1], currentIndex, threadIndex) == currentIndex) {
				indices[index] = currentIndex;
				swap(nodes[index], nodes[split.index+1]);
				break;
			}
		}
		else
			break;
	}
}

__global__ static void swapMaxTriangleTPS(const float* keys,
	const unsigned int pendingSplits, const Split* splits,
	unsigned int* indices, unsigned int* nodes)
{
	unsigned int index = blockDim.x * blockIdx.x + threadIdx.x;
	if(index >= pendingSplits)
		return;

	const Split split     = splits[index];
	float maxValue        = keys[indices[split.index+1]];
	unsigned int maxIndex = split.index+1;

	for(unsigned int i=split.index+2; i<split.index+split.size; i++) {
		const float value = keys[indices[i]];
		if(value > maxValue) {
			maxIndex = i;
			maxValue = value;
		}
	}

	if(maxIndex != split.index+1) {
		swap(indices[split.index+1], indices[maxIndex]);
		swap(nodes[split.index+1], nodes[maxIndex]);
	}
}

__global__ static void updateNodesTPT(const unsigned int count,
	const unsigned int pendingSplits, const Split* splits,
	const unsigned int doneNodes, unsigned int* nodes)
{
	const unsigned int index = blockDim.x * blockIdx.x + threadIdx.x + 2*doneNodes;
	if(index >= count)
		return;

	Split split;
	const unsigned int splitIndex = findSplit(index, pendingSplits, splits, split);
	const unsigned int nodeIndex  = doneNodes + splitIndex;

	if(index < split.index+2)
		nodes[index] = nodeIndex;
	else
		nodes[index] = 2*nodeIndex+1;
}

__global__ static void updateNodesTPS(
	const unsigned int pendingSplits, const Split* splits,
	const unsigned int doneNodes, unsigned int* nodes)
{
	const unsigned int index = blockDim.x * blockIdx.x + threadIdx.x;
	if(index >= pendingSplits)
		return;

	const Split split = splits[index];
	const unsigned int nodeIndex = doneNodes + index;

	nodes[split.index]   = nodeIndex;
	nodes[split.index+1] = nodeIndex;
	for(unsigned int i=split.index+2; i<split.index + split.size; i++) {
		nodes[i] = 2*nodeIndex+1;
	}
}

__global__ static void emitSplitsKernel(const unsigned int pendingSplits,
	const Split* inSplits, Split* outSplits, unsigned int* generatedSplits)
{
	const unsigned int index = blockDim.x * blockIdx.x + threadIdx.x;
	if(index >= pendingSplits)
		return;

	Split split = inSplits[index];
	if(split.size <= 2)
		return;

	unsigned int newSplits = 0;
	unsigned int numL, numR, pL, pR;
	calcPartition(split.size, numL, numR);
	pL = split.index + 2 * (pendingSplits - index);
	pR = pL + numL;

	outSplits[2*index].index = pL;
	outSplits[2*index].size  = numL;
	if(numL > 0) newSplits++;

	outSplits[2*index+1].index = pR;
	outSplits[2*index+1].size  = numR;
	if(numR > 0) newSplits++;

	atomicAdd(generatedSplits, newSplits);
}

__global__ static void applyIndices(const Geometry source, Geometry dest, const unsigned int* indices)
{
	const unsigned int index = blockDim.x * blockIdx.x + threadIdx.x;
	if(index >= source.count)
		return;

	Primitive buffer;
	const unsigned int position = indices[index];

	// Copy vertices
	buffer.readValues(source.vertices + position * Geometry::TriangleParams);
	buffer.writeValues(dest.vertices + index * Geometry::TriangleParams);

	// Copy normals
	buffer.readValues(source.normals + position * Geometry::TriangleParams);
	buffer.writeValues(dest.normals + index * Geometry::TriangleParams);
}

__host__ static void emitSplits(const unsigned int pendingSplits,
	Split* inSplits, Split* outSplits, unsigned int* hptrGeneratedSplits, unsigned int* dptrGeneratedSplits)
{
	dim3 blockSize(256);
	dim3 gridSize = make_grid(blockSize, dim3(pendingSplits));

	*hptrGeneratedSplits = 0;
	emitSplitsKernel<<<gridSize, blockSize>>>(pendingSplits, inSplits, outSplits, dptrGeneratedSplits);
	hipDeviceSynchronize();

	thrust::device_ptr<Split> ptrInSplits(inSplits);
	thrust::device_ptr<Split> ptrOutSplits(outSplits);
	thrust::copy_if(ptrOutSplits, ptrOutSplits + (*hptrGeneratedSplits), ptrInSplits, ActiveSplitPredicate());
}

__host__ static void sortTriangles(const unsigned int count,
	const float* keys, unsigned int* indices, unsigned int* nodes,
	thrust::device_vector<unsigned int>& permutation,
	thrust::device_vector<unsigned int>& tempi, thrust::device_vector<float>& tempf)
{
	thrust::device_ptr<const float>  ptrKeys(keys);
	thrust::device_ptr<unsigned int> ptrIndices(indices);
	thrust::device_ptr<unsigned int> ptrNodes(nodes);

	// Generate identity permutation
	thrust::sequence(permutation.begin(), permutation.end(), 0, 1);

	// Spatial sort
	thrust::gather(ptrIndices, ptrIndices + count, ptrKeys, tempf.begin());
	thrust::stable_sort_by_key(tempf.begin(), tempf.end(), permutation.begin());

	// Node sort
	thrust::gather(permutation.begin(), permutation.end(), ptrNodes, tempi.begin());
	thrust::stable_sort_by_key(tempi.begin(), tempi.end(), permutation.begin());

	// Apply permutations to indices
	thrust::copy(ptrIndices, ptrIndices + count, tempi.begin());
	thrust::gather(permutation.begin(), permutation.end(), tempi.begin(), ptrIndices);

	// Apply permutations to nodes
	thrust::copy(ptrNodes, ptrNodes + count, tempi.begin());
	thrust::gather(permutation.begin(), permutation.end(), tempi.begin(), ptrNodes);
}

__host__ bool cudaRebuildNMH(Geometry& geometry)
{
	const size_t numLevels = log2i(geometry.count / 2) + 1;
	const size_t maxSplits = 1 << numLevels;

	// Result geometry
	Geometry result;
	result.initialize();

	// Prefer L1 cache
	hipDeviceSetCacheConfig(hipFuncCachePreferL1);

	// Triangle indices
	unsigned int* indices;
	hipMalloc(&indices, geometry.count * sizeof(unsigned int));
	thrust::device_ptr<unsigned int> thrustIndices(indices);
	thrust::sequence(thrustIndices, thrustIndices + geometry.count);
	
	// Node indices
	unsigned int* nodes;
	hipMalloc(&nodes, geometry.count * sizeof(unsigned int));
	hipMemset(nodes, 0, geometry.count * sizeof(unsigned int));

	// Active splits
	Split *inSplits, *outSplits;
	hipMalloc(&inSplits, 2 * maxSplits * sizeof(Split));
	hipMalloc(&outSplits, 2 * maxSplits * sizeof(Split));

	const Split initialSplit(0, geometry.count);
	hipMemcpy(inSplits, &initialSplit, sizeof(Split), hipMemcpyHostToDevice);

	// Vertices bounds
	float* verticesMin[3];
	float* verticesMax[3];
	for(int i=0; i<3; i++) {
		hipMalloc(&verticesMin[i], geometry.count * sizeof(float));
		hipMalloc(&verticesMax[i], geometry.count * sizeof(float));
	}

	// GPU generated split count
	unsigned int* hptrGeneratedSplits;
	unsigned int* dptrGeneratedSplits;
	hipHostAlloc(&hptrGeneratedSplits, sizeof(unsigned int), hipHostMallocMapped);
	hipHostGetDevicePointer(&dptrGeneratedSplits, hptrGeneratedSplits, 0);

	// Temporary arrays for sorting
	thrust::device_vector<unsigned int> permutation(geometry.count);
	thrust::device_vector<unsigned int> tempi(geometry.count);
	thrust::device_vector<float> tempf(geometry.count);

	// Computation state
	int axis = 0;
	unsigned int pendingSplits = 1;
	unsigned int doneNodes     = 0;

	dim3 gridSize, blockSize;

	// Compute triangle bounds
	blockSize = dim3(192);
	gridSize  = make_grid(blockSize, dim3(geometry.count));
	computeTriangleBounds<<<gridSize, blockSize>>>(geometry,
		verticesMin[0], verticesMin[1], verticesMin[2],
		verticesMax[0], verticesMax[1], verticesMax[2]);

	// Loop every level of the hierarchy
	for(unsigned int i=0; i<numLevels; i++) {
		// 1. Lexicographical sort
		const float* keysMin = verticesMin[axis];
		const float* keysMax = verticesMax[axis];

		sortTriangles(geometry.count, keysMin, indices, nodes, permutation, tempi, tempf);

		// Finish if on last level
		if(i == numLevels-1)
			break;

		// 2. Find maximal triangle in every split
		blockSize = dim3(256);
		if(pendingSplits <= REBUILDER_TPS_THRESHOLD) {
			gridSize = make_grid(blockSize, dim3(geometry.count - 2*doneNodes));
			swapMaxTriangleTPT<<<gridSize, blockSize>>>(geometry.count, keysMax, pendingSplits, inSplits,
				indices, doneNodes, nodes);
		}
		else {
			gridSize = make_grid(blockSize, dim3(pendingSplits));
			swapMaxTriangleTPS<<<gridSize, blockSize>>>(keysMax, pendingSplits, inSplits, indices, nodes);
		}

		// 3. Update node values
		blockSize = dim3(256);
		if(pendingSplits <= REBUILDER_TPS_THRESHOLD) {
			gridSize = make_grid(blockSize, dim3(geometry.count - 2*doneNodes));
			updateNodesTPT<<<gridSize, blockSize>>>(geometry.count, pendingSplits, inSplits, doneNodes, nodes);
		}
		else {
			gridSize = make_grid(blockSize, dim3(pendingSplits));
			updateNodesTPS<<<gridSize, blockSize>>>(pendingSplits, inSplits, doneNodes, nodes);
		}

		if(i < max(0, int(numLevels-2))) {
			// 4. Emit new splits
			emitSplits(pendingSplits, inSplits, outSplits, hptrGeneratedSplits, dptrGeneratedSplits);

			doneNodes    += pendingSplits;
			pendingSplits = *hptrGeneratedSplits;
		}

		// 5. Increment axis
		axis = (axis + 1) % 3;
	}

	// Free resources
	hipFree(nodes);
	hipFree(inSplits);
	hipFree(outSplits);
	hipHostFree(hptrGeneratedSplits);

	for(int i=0; i<3; i++) {
		hipFree(verticesMin[i]);
		hipFree(verticesMax[i]);
	}

	// Apply indices
	result.resize(geometry.count, Geometry::AllocDefault);

	blockSize = dim3(256);
	gridSize  = make_grid(blockSize, dim3(geometry.count));
	applyIndices<<<gridSize, blockSize>>>(geometry, result, indices);
	hipFree(indices);

	geometry.free();
	geometry = result;
	
	// Restore default cache setting
	hipDeviceSetCacheConfig(hipFuncCachePreferNone);
	return true;
}