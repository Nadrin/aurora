#include "hip/hip_runtime.h"
/* Aurora Renderer
 * Copyright (c) 2013 Michal Siejak
 * Licensed under MIT open-source license, see COPYING.txt file for details.
 */

#include <stdafx.h>
#include <kernels/kernels.h>

using namespace Aurora;

#include <kernels/lib/common.cuh>
#include <kernels/lib/intersect.cuh>
#include <kernels/lib/light.cuh>

__global__ static void cudaRaycastKernel(const Geometry geometry, const ShadersArray shaders, const LightsArray lights,
	const unsigned int numRays, Ray* rays, float4* pixels)
{
	unsigned int threadId = blockDim.x * blockIdx.x + threadIdx.x;
	if(threadId >= numRays)
		return;

	float3 color = make_float3(0.0f, 0.0f, 0.0f);

	unsigned int triangleIndex;
	Ray ray = rays[threadId];
	ray.t   = Infinity;

	if(intersect(geometry, ray, triangleIndex)) {
		float3 N, T, S;
		getBasisVectors(geometry, triangleIndex, ray.u, ray.v, N, S, T);

		const float3 P = ray.point();

		const unsigned int shaderID = getSafeID(geometry.shaders[triangleIndex]);
		const Shader shader = shaders[shaderID];
		
		color = shader.ambientColor;
		for(unsigned int i=0; i<lights.size; i++) {
			//const float3 L    = worldToLocal(lights[i].getL(P), N, S, T);
			const float3 L = make_float3(0.0f, 0.0f, -1.0f);
			const float dotNL = cosTheta(L);
			if(dotNL > 0.0f)
				color = color + dotNL * shader.diffuse * lights[i].intensity * shader.color * lights[i].color;
		}
	}

	pixels[ray.id] = make_float4(
		clamp(color.x, 0.0f, 1.0f),
		clamp(color.y, 0.0f, 1.0f),
		clamp(color.z, 0.0f, 1.0f),
		1.0f);
}

void cudaRaycast(const Geometry& geometry, const ShadersArray& shaders, const LightsArray& lights,
	const unsigned int numRays, Ray* rays, void* pixels)
{
	dim3 blockSize(256);
	dim3 gridSize = make_grid(blockSize, dim3(numRays));
	cudaRaycastKernel<<<gridSize, blockSize>>>(geometry, shaders, lights, numRays, rays, (float4*)pixels);
}